// Uses cuRAND to generate a normal distribution of numbers

// Include necessary files
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <stdio.h>

int main(void) {
  // Allocate pointers for host and device memory
    float *h_output;
    float *d_output;

  // Determine number of random numbers using user input
    int number_of_random_nums;
    printf("Enter the number of random numbers to generate: ");
    scanf("%d", &number_of_random_nums);

  // malloc() host memory (in RAM)
    
