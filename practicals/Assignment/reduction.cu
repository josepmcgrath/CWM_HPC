#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// Include cuda functions
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

// Include timing functions
#include <sys/time.h>
#define MILLION 1000000.0

#define MAX_THREADS 1024

__global__ void reduction(float *d_input, float *d_output, int num_els)
{
    // Allocate shared memory

    __shared__  float smem_array[MAX_THREADS];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // first, each thread loads data into shared memory
    if (index < num_els) {
      smem_array[tid] = d_input[index];
    } else {
      smem_array[tid] = 0;
    }
    // next, we perform binary tree reduction

    for (int d = blockDim.x/2; d > 0; d /= 2) {
      __syncthreads();  // ensure previous step completed 
      if (tid<d)  smem_array[tid] += smem_array[tid+d];
    }

    // finally, first thread puts result into global memory

    if (tid==0) d_output[blockIdx.x] = smem_array[0];
}

double wall_clock_time (void) {

  double secs;
  struct timeval tp;

  gettimeofday (&tp,NULL);
  secs = (MILLION * (double) tp.tv_sec + (double) tp.tv_usec) / MILLION;
  return secs;

}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int main( int argc, const char** argv) 
{
    int num_els, num_threads, mem_size, num_blocks;

    float *h_data;
    float *d_input, *d_output;
    
    // timing variables
     
    double time_start, time_end;
    
    // Take user input for number of random numbers
    
    printf("Enter a number of random numbers: ");
    scanf("%d", &num_els);

    // initialise card

    num_threads = MAX_THREADS;
    mem_size    = sizeof(float) * num_els;
    num_blocks = ((num_els % MAX_THREADS) == 0) ? (num_els / MAX_THREADS) : (num_els / MAX_THREADS + 1);
    
    // allocate host memory to store the input data
    // and initialize to integer values between 0 and 1000

    h_data = (float*) malloc(mem_size);
      
    // allocate device memory input and output arrays

    hipMalloc((void**)&d_input, mem_size);
    hipMalloc((void**)&d_output, (num_blocks * sizeof(float)));

    // Use cuRAND to generate input data

      // Create pseudo-random number generator
        hiprandGenerator_t gen;
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

      // Set the generator options
        hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

      // Generate the randoms
        hiprandGenerateNormal(gen, d_input, num_els, 0.0f, 1.0f);


    // copy host memory to device input array

   // cudaMemcpy(d_input, h_data, mem_size, cudaMemcpyHostToDevice);

    // execute the kernel
    
      // start time
      time_start = wall_clock_time ( );

    reduction<<<num_blocks,num_threads>>>(d_input,d_output, num_els);

      // end time
       time_end = wall_clock_time ( );

    // copy result from device to host

    hipMemcpy(h_data, d_output, num_blocks * sizeof(float), hipMemcpyDeviceToHost);
    
    // Sum results from all blocks
    
    for (int i = 1; i < num_blocks; i++) {
      h_data[0] += h_data[i];
    }

    // check results
    printf(" process time      = %e s\n", time_end - time_start);
    printf("Total sum: %f\n", h_data[0]);
    printf("reduction error = %f\n",h_data[0]/num_els);

    // cleanup memory

    free(h_data);
    hipFree(d_input);
    hipFree(d_output);
    hiprandDestroyGenerator(gen);
    
    // CUDA exit -- needed to flush printf write buffer

    hipDeviceReset();
}

